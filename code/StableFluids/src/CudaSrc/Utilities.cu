#include "hip/hip_runtime.h"
#include "utilities.cuh"
#include "Fluid.cuh"
#include <math.h>


namespace StableFluidsCuda {

	__device__ void set_corner_gpu(float* x, int N)
	{
		//printf(__FUNCTION__);
		x[IX(0, 0)] = 0.5 * (x[IX(1, 0)] + x[IX(0, 1)]);
		x[IX(0, N - 1)] = 0.5 * (x[IX(1, N - 1)] + x[IX(0, N - 2)]);
		x[IX(N - 1, 0)] = 0.5 * (x[IX(N - 2, 0)] + x[IX(N - 1, 1)]);
		x[IX(N - 1, N - 1)] = 0.5 * (x[IX(N - 2, N - 1)] + x[IX(N - 1, N - 2)]);
	}

	__device__ void set_bnd_gpu(int b, float* x, int N, int tid)
	{			
		int i = tid % N;
		int j = tid / N;
		if (i >= 1 && i < N - 1 && j >= 1 && j < N - 1)
		{
			
			x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
			x[IX(i, N - 1)] = b == 2 ? -x[IX(i, N - 2)] : x[IX(i, N - 2)];
			x[IX(0, j)] = b == 1 ? -x[IX(1, j)] : x[IX(1, j)];
			x[IX(N - 1, j)] = b == 1 ? -x[IX(N - 2, j)] : x[IX(N - 2, j)];
		}
		__syncthreads();
		int n2 = N / 2;
		if (i == n2 && j == n2) {
			set_corner_gpu(x, N);
			return;
		}		
	}

	__device__ void lin_solve_gpu(int b, float* x, float* x0, float a, float c, int iter, int N, int tid)
	{
		int localID = threadIdx.x;		
		int i = tid % N;
		int j = tid / N;

		__shared__ float local_x[NUM_THREADS];

		if (i < 1 || i > N - 2) return;
		if (j < 1 || j > N - 2) return;

		float cRecip = 1.0 / c;
		for (int k = 0; k < iter; k++) {
			local_x[localID] =
				(x0[IX(i, j)]
					+ a * (x[IX(i + 1, j)]
						+ x[IX(i - 1, j)]
						+ x[IX(i, j + 1)]
						+ x[IX(i, j - 1)]
						)) * cRecip;
			__syncthreads();
			x[IX(i, j)] = local_x[localID];
			__syncthreads();
			set_bnd_gpu(b, x, N, tid);

		}
	}

	__global__ void diffuse_gpu(int b, float* x, float* x0, float diff, float dt, int iter, int N)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid >= N * N) return;

		float a = dt * diff * (N - 2) * (N - 2);
		lin_solve_gpu(b, x, x0, a, 1 + 6 * a, iter, N, tid); 	
		
	}

	__global__ void diffuse_gpu_linear(int b, float* x, float* x0, float diff, float dt, int iter, int N)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid >= N * N) return;

		float a = dt * diff * (N - 2) * (N - 2);
		//lin_solve_gpu(b, x, x0, a, 1 + 6 * a, iter, N, tid);
		float c = 1 + 6 * a;
		float cRecip = 1.0 / c;
		//*************************************
	  //  iter = 1;
		for (int k = 0; k < iter; k++) {
			for(int i = 0; i < N*N; i++)
			{
				lin_solve_gpu(b, x, x0, a, 1 + 6 * a, iter, N, i);
			}
			/*for (int j = 1; j < N - 1; j++) {
				for (int i = 1; i < N - 1; i++) {
					x[IX(i, j)] =
						(x0[IX(i, j)]
							+ a * (x[IX(i + 1, j)]
								+ x[IX(i - 1, j)]
								+ x[IX(i, j + 1)]
								+ x[IX(i, j - 1)]
								)) * cRecip;
				}
			}*/
			for (int i = 0; i < N * N; i++)
			{
				set_bnd_gpu(b, x, N, i);

			}

			//set_bnd(b, x, N);
			/*for (int i = 1; i < N - 1; i++) {
				x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
				x[IX(i, N - 1)] = b == 2 ? -x[IX(i, N - 2)] : x[IX(i, N - 2)];
				x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
				x[IX(N - 1, i)] = b == 1 ? -x[IX(N - 2, i)] : x[IX(N - 2, i)];
			}

			x[IX(0, 0)] = 0.5 * (x[IX(1, 0)] + x[IX(0, 1)]);
			x[IX(0, N - 1)] = 0.5 * (x[IX(1, N - 1)] + x[IX(0, N - 2)]);
			x[IX(N - 1, 0)] = 0.5 * (x[IX(N - 2, 0)] + x[IX(N - 1, 1)]);
			x[IX(N - 1, N - 1)] = 0.5 * (x[IX(N - 2, N - 1)] + x[IX(N - 1, N - 2)]);*/
		}


	}

	__global__ void project_gpu(float* velocX, float* velocY, float* p, float* div, int iter, int N)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid >= N * N) return;

		int i = tid % N;
		int j = tid / N;

		if (i < 1 || i > N - 2) return;
		if (j < 1 || j > N - 2) return;


		div[IX(i, j)] = -0.5f * (
			velocX[IX(i + 1, j)]
			- velocX[IX(i - 1, j)]
			+ velocY[IX(i, j + 1)]
			- velocY[IX(i, j - 1)]
			) / N;
		p[IX(i, j)] = 0;
		__syncthreads();

		set_bnd_gpu(0, div, N, tid);
		__syncthreads();

		set_bnd_gpu(0, p, N, tid);
		__syncthreads();

		lin_solve_gpu(0, p, div, 1, 6, iter, N, tid);
		__syncthreads();


		velocX[IX(i, j)] -= 0.5f * (p[IX(i + 1, j)]
			- p[IX(i - 1, j)]) * N;
		velocY[IX(i, j)] -= 0.5f * (p[IX(i, j + 1)]
			- p[IX(i, j - 1)]) * N;
		__syncthreads();

		set_bnd_gpu(1, velocX, N, tid);
		__syncthreads();

		set_bnd_gpu(2, velocY, N, tid);
		//__syncthreads();

	}

	__global__ void advect_gpu(int b, float* d, float* d0, float* velocX, float* velocY, float dt, int N)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid >= N * N) return;

		int i = tid % N;
		int j = tid / N;

		if (i < 1 || i > N - 2) return;
		if (j < 1 || j > N - 2) return;

		float i0, i1, j0, j1;

		float dtx = dt * (N - 2);
		float dty = dt * (N - 2);

		float s0, s1, t0, t1;
		float tmp1, tmp2, x, y;

		float Nfloat = N;
		float ifloat = i;
		float jfloat = j;

		tmp1 = dtx * velocX[tid];
		tmp2 = dty * velocY[tid];
		x = ifloat - tmp1;
		y = jfloat - tmp2;

		if (x < 0.5f) x = 0.5f;
		if (x > Nfloat + 0.5f) x = Nfloat + 0.5f;
		i0 = floorf(x);
		i1 = i0 + 1.0f;
		if (y < 0.5f) y = 0.5f;
		if (y > Nfloat + 0.5f) y = Nfloat + 0.5f;
		j0 = floorf(y);
		j1 = j0 + 1.0f;

		s1 = x - i0;
		s0 = 1.0f - s1;
		t1 = y - j0;
		t0 = 1.0f - t1;

		int i0i = i0;
		int i1i = i1;
		int j0i = j0;
		int j1i = j1;

		d[tid] =
			s0 * (t0 * d0[IX(i0i, j0i)] + t1 * d0[IX(i0i, j1i)])
			+ s1 * (t0 * d0[IX(i1i, j0i)] + t1 * d0[IX(i1i, j1i)]);
		__syncthreads();

		set_bnd_gpu(b, d, N, tid);
		//__syncthreads();

	}
}